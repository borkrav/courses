#include "hip/hip_runtime.h"
#include "master.h"
#include "agent.h"
#include "fitness.h"
#include <iostream>
#include <thread>
#include <mutex>
#include <condition_variable> 
#include <iostream>
#include <atomic>

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <math.h>

using namespace std;

const int THREADS = 1;

mutex mtx;
condition_variable cv;
condition_variable cv2;


bool ready[THREADS] = { false };

bool waker = true;
bool done = false;

atomic<int> counter;


using namespace std;


void work(agent** particles, int numParticles, float(*func)(int dim, float *), int id){

	unique_lock<mutex> lck(mtx);

	while (true){

		while (!ready[id]) cv.wait(lck);
		lck.unlock();
		if (done) break;

		for (int j = id; j < numParticles; j += THREADS){
			particles[j]->runStep(func);
		}

		lck.lock();

		ready[id] = false;


		if (++counter == THREADS){
			counter = 0;
			waker = true;
			cv2.notify_all();
		}

	}

}


__global__ void calculateFitness(int *dim, float *pos, float *fitness, int *numParticles){

	int particle = blockIdx.x * blockDim.x + threadIdx.x;

	if (particle < *numParticles){

		float total = 0;

		//sphere function
		/*
		for (int i = 0; i < *dim; i++){

			float val = pos[particle * (*dim) + i];
			
			total += (pow(val, 2));

		}
		*/

		//Rosenbrock function

		/*
		for (int i = 0; i < *dim - 1; i++){


			total += 100 * pow((pos[particle * (*dim) + i + 1] - pow(pos[particle * (*dim) + i], 2)), 2) + pow((pos[particle * (*dim) + i] - 1), 2);

		}
		*/
		/*
		float sum1 = 0;

		for (int i = 0; i < *dim; i++){
			float val = pos[particle * (*dim) + i];

			sum1 += pow(val, 2);
		}

		float sum2 = 0;

		for (int i = 0; i < *dim; i++){
			float val = pos[particle * (*dim) + i];

			sum2 += cos(2 * (atan(1.0) * 4)*val);
		}

		total = -20 * expf(-0.2*sqrt(1 / *dim*sum1)) - expf(1 / *dim*sum2) + 20 * expf(1.0);

		*/



		for (int i = 0; i < *dim; i++){

			float val = pos[particle * (*dim) + i];

			total += 2 * pow(val, 2) * sin(exp(sin(exp(sin(val)))));

		}



		fitness[particle] = total;


	}



}



void master::finishStep(int numParticles, int dim){

	float *dev_pos;
	float *dev_fitness;
	int *dev_dim;
	int *dev_numParticles;

	int cudastatus;

	float *h_pos = new float[numParticles*dim];
	for (int i = 0; i < numParticles; i++){
		for (int j = 0; j < dim; j++){
			h_pos[(i*dim)+j] = particles[i]->pos[j];
		}
	}

	//for (int i = 0; i < numParticles*dim; i++){
	//	cout << h_pos[i] << endl;
	//}

	float *h_fitness = new float[numParticles]{0};

	cudastatus = hipMalloc((void**)&dev_pos, numParticles * dim * sizeof(float));
	cudastatus = hipMalloc((void**)&dev_fitness, numParticles * sizeof(float));
	cudastatus = hipMalloc((void**)&dev_dim, sizeof(int));
	cudastatus = hipMalloc((void**)&dev_numParticles, sizeof(int));
	
	cudastatus = hipMemcpy(dev_pos, h_pos, numParticles * dim * sizeof(float), hipMemcpyHostToDevice);
	cudastatus = hipMemcpy(dev_fitness, h_fitness, numParticles * sizeof(float), hipMemcpyHostToDevice);
	cudastatus = hipMemcpy(dev_dim, &dim, sizeof(int), hipMemcpyHostToDevice);
	cudastatus = hipMemcpy(dev_numParticles, &numParticles, sizeof(int), hipMemcpyHostToDevice);

	if (numParticles > 32){
		calculateFitness << < ceil((float)numParticles / 32), 32 >> >(dev_dim, dev_pos, dev_fitness, dev_numParticles);
	}

	else {
		calculateFitness << < 1, numParticles >> >(dev_dim, dev_pos, dev_fitness, dev_numParticles);
	}

	hipDeviceSynchronize();

	//run kernel here

	hipMemcpy(h_fitness, dev_fitness, numParticles * sizeof(float), hipMemcpyDeviceToHost);

	//for (int i = 0; i < numParticles; i++)
	//	cout << h_fitness[i];

	float gBest = gBest::getInstance().fitness;

	for (int i = 0; i < numParticles; i++){
		if (MAXIMIZE){
			if (h_fitness[i] > gBest){
				gBest::getInstance().fitness = h_fitness[i];

				for (int j = 0; j < dim; j++){
					gBest::getInstance().pos[j] = particles[i]->pos[j];
				}
			}

			if (h_fitness[i] > particles[i]->fitness){
				particles[i]->fitness = h_fitness[i];

				for (int j = 0; j < dim; j++){
					particles[i]->pbest[j] = particles[i]->pos[j];
				}
			}
		}

		else{
			if (h_fitness[i] < gBest){
				gBest::getInstance().fitness = h_fitness[i];
				//cout << gBest::getInstance().fitness << endl;
				for (int j = 0; j < dim; j++){
					gBest::getInstance().pos[j] = particles[i]->pos[j];
				}
			}
			if (h_fitness[i] < particles[i]->fitness){
				particles[i]->fitness = h_fitness[i];

				for (int j = 0; j < dim; j++){
					particles[i]->pbest[j] = particles[i]->pos[j];
				}
			}
		}

	}

	hipFree(dev_pos);
	hipFree(dev_dim);
	hipFree(dev_fitness);
	hipFree(dev_numParticles);

	delete[] h_pos;
	delete[] h_fitness;



}


master::master(int numParticles, int dim, int runs, float(*func)(int dim, float *)){

	particles = new agent*[numParticles];

	for (int i = 0; i < numParticles; i++){
		particles[i] = new agent(dim, func);
	}


	gBest::getInstance().initArray(dim);


	thread threads[THREADS];

	for (int i = 0; i < THREADS; ++i){
		threads[i] = thread(work, particles, numParticles, func, i);
	}

	unique_lock<mutex> lck(mtx);

	for (int i = 0; i < runs; i++){

		for (int i = 0; i < THREADS; i++) ready[i] = true;

		//finish step
		finishStep(numParticles, dim);

		cv.notify_all();

		waker = false;

		while (!waker) cv2.wait(lck);

	}

	done = true;
	cout << "done" << endl;
	for (int i = 0; i < THREADS; i++) ready[i] = true;
	cv.notify_all();
	lck.unlock();

	for (auto& th : threads) th.join();
	

	//output gbest - optimal solution found
	for (int i = 0; i < dim; i++){
		cout << gBest::getInstance().pos[i] << endl;
	}


}


master::~master(){

	for (int i = 0; i < dim; i++){
		delete particles[i];
	}

	delete[] particles;




}
